#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#include "cuda_utils.cuh"
#include "device_rng.cuh"

__device__ hiprandState *d_randStates = nullptr;
__device__ int d_numStates = 0;

// initiate random states
// use the same random seed, different sub-sequence, and with offset of 0
__global__ void initRandStates(unsigned long long int random_seed, int totalStates)
{
    // get thread idx
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ind < totalStates){
        hiprand_init(random_seed, ind, 0, &d_randStates[ind]);

        // todo: add the following block to debug 
        if (ind < 2)
        {
            printf("temp check rng init.\n");
            printf("Thread %d: d = %u, v0 = %u, boxmuller_flag = %d, boxmuller_extra = %f\n",
                   ind, d_randStates[ind].d, d_randStates[ind].v[0],
                   d_randStates[ind].boxmuller_flag, d_randStates[ind].boxmuller_extra);
        }
    }
}

// Function to initialize RNG states.
void init_rand_states(unsigned long long int seed, int numStates)
{
    CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_numStates), &numStates, sizeof(int), 0, hipMemcpyHostToDevice));

    // todo: add the following block to debug
    hiprandState *checkPtr0 = nullptr;
    CALL_CUDA(hipMemcpyFromSymbol(&checkPtr0, HIP_SYMBOL(d_randStates), sizeof(checkPtr0), 0, hipMemcpyDeviceToHost));
    printf("init device pointer = %p\n", checkPtr0);

    hiprandState *tmpPtr = nullptr;
    CALL_CUDA(hipMalloc((void **)&tmpPtr, numStates * sizeof(hiprandState)));
    CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_randStates), &tmpPtr, sizeof(tmpPtr), 0, hipMemcpyHostToDevice));
    tmpPtr = nullptr;

    // todo: add the following block to debug (verify device pointer has been updated successfully)
    hiprandState *checkPtr = nullptr;
    CALL_CUDA(hipMemcpyFromSymbol(&checkPtr, HIP_SYMBOL(d_randStates), sizeof(checkPtr), 0, hipMemcpyDeviceToHost));
    printf("updated device pointer = %p\n", checkPtr);

    // define kernel grids
    int threadsPerBlock = 256;
    int blocks = (numStates + threadsPerBlock - 1) / threadsPerBlock;

    // launch kernel function
    initRandStates<<<blocks, threadsPerBlock>>>(seed, numStates);
    CALL_CUDA(hipGetLastError());
    hipDeviceSynchronize();
}

void free_rand_states()
{
    // copy device pointer/variable to the host
    hiprandState *h_randStates = nullptr;
    int h_numStates = 0;
    CALL_CUDA(hipMemcpyFromSymbol(&h_randStates, HIP_SYMBOL(d_randStates), sizeof(d_randStates), 0, hipMemcpyDeviceToHost));
    CALL_CUDA(hipMemcpyFromSymbol(&h_numStates, HIP_SYMBOL(d_numStates), sizeof(int), 0, hipMemcpyDeviceToHost));
    if (h_randStates){
        CALL_CUDA(hipFree(h_randStates));
        h_randStates = nullptr;
        CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_randStates), &h_randStates, sizeof(h_randStates), 0, hipMemcpyHostToDevice));
    }

    if (h_numStates != 0){
        h_numStates = 0;
        CALL_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_numStates), &h_numStates, sizeof(int), 0, hipMemcpyHostToDevice));
    }
}
