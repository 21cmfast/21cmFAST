#include "hip/hip_runtime.h"
// Most of the following includes likely can be removed.
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <complex.h>
#include <fftw3.h>

// GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// We use thrust for reduction
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/functional.h> // thrust::plus

#include "cexcept.h"
#include "exceptions.h"
#include "logger.h"
#include "Constants.h"
#include "indexing.h"
#include "InputParameters.h"
#include "OutputStructs.h"
#include "heating_helper_progs.h"
#include "elec_interp.h"
#include "interp_tables.h"
#include "debugging.h"
#include "cosmology.h"
#include "hmf.h"
#include "dft.h"
#include "filtering.h"
#include "thermochem.h"
#include "interpolation.h"

#include "SpinTemperatureBox.h"
#include "cuda_utils.cuh"


__device__ inline double EvaluateRGTable1D_f_gpu(double x, double x_min, double x_width, float *y_arr) {

    int idx = (int)floor((x - x_min) / x_width);

    double table_val = x_min + x_width * (float)idx;
    double interp_point = (x - table_val) / x_width;

    return y_arr[idx] * (1 - interp_point) + y_arr[idx + 1] * (interp_point);
}

__global__ void compute_sfrd_kernel(
    double x_min, // reference
    double x_width, // reference
    float *y_arr, // reference
    float *dens_R_grid, // reference
    double zpp_growth_R_ct, // value
    float *sfrd_grid, // star formation rate density grid to be updated
    double *fcoll_tmp, // temp buffer for later summation
    unsigned long long num_pixels // length of input data
) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    double curr_dens;
    double fcoll;

   // Bound check (in case number of threads != multiple of block size)
    if (idx >= num_pixels) {
        return;
    }
    // Compute current density from density grid value * redshift-scaled growth factor
    curr_dens = dens_R_grid[idx] * zpp_growth_R_ct;

    // Compute fraction of mass that has collapsed to form stars/other structures
    fcoll = exp(EvaluateRGTable1D_f_gpu(curr_dens, x_min, x_width, y_arr));

    // Update the fcoll temp array
    fcoll_tmp[idx] = fcoll;

    // Update the relevant cells in the star formation rate density grid
    sfrd_grid[idx] = (1. + curr_dens) * fcoll;
}

void init_sfrd_gpu_data_simple(
    float *dens_R_grid, // input data
    float *sfrd_grid, // star formation rate density grid to be updated
    unsigned long long num_pixels, // length of input data
    unsigned int nbins, // nbins for sfrd_grid->y
    float **d_y_arr, // copies of pointers to pointers
    float **d_dens_R_grid,
    float **d_sfrd_grid,
    double **d_fcoll_tmp
) {
    // Allocate device memory
    CALL_CUDA(hipMalloc(d_y_arr, sizeof(float) * nbins)); // already pointers to pointers (no & needed)
    CALL_CUDA(hipMalloc(d_dens_R_grid, sizeof(float) * num_pixels)); // TODO: check removal of (void**) since already pointers to pointers, so cast is redundant
    CALL_CUDA(hipMalloc(d_sfrd_grid, sizeof(float) * num_pixels));
    CALL_CUDA(hipMalloc(d_fcoll_tmp, sizeof(double) * num_pixels));
    LOG_INFO("SFRD_conditional_table.y_arr, density and sfrd grids, and fcoll temp array allocated on device.");

    // Initialise sfrd_grid to 0 (fill with byte=0)
    CALL_CUDA(hipMemset(*d_sfrd_grid, 0, sizeof(float) * num_pixels)); // dereference the pointer to a pointer (*)
    CALL_CUDA(hipMemset(*d_fcoll_tmp, 0, sizeof(double) * num_pixels));
    LOG_INFO("sfrd grid and fcoll temp array initialised to 0.");
}

double calculate_sfrd_gpu_simple(
    RGTable1D_f *SFRD_conditional_table, // input data
    float *dens_R_grid, // input data
    double *zpp_growth, // input data
    int R_ct, // filter step/loop iteration/spherical annuli (out of 40 by default)
    float *sfrd_grid, // star formation rate density grid to be updated
    unsigned long long num_pixels, // length of input data
    float *d_y_arr,
    float *d_dens_R_grid,
    float *d_sfrd_grid,
    double *d_fcoll_tmp
) {
    // Get growth factor for current filter step
    double zpp_growth_R_ct = zpp_growth[R_ct];

    // Copy data from host to device
    CALL_CUDA(hipMemcpy(d_y_arr, SFRD_conditional_table->y_arr, sizeof(float) * SFRD_conditional_table->n_bin, hipMemcpyHostToDevice));
    CALL_CUDA(hipMemcpy(d_dens_R_grid, dens_R_grid, sizeof(float) * num_pixels, hipMemcpyHostToDevice));
    LOG_INFO("SFRD_conditional_table.y_arr and density grid copied to device.");

    // Get max threads/block for device
    int maxThreadsPerBlock;
    CALL_CUDA(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));

    // Set threads/block based on device max
    unsigned int threadsPerBlock;
    if (maxThreadsPerBlock >= 512) {
        threadsPerBlock = 512;
    } else if (maxThreadsPerBlock >= 256) {
        threadsPerBlock = 256;
    } else if (maxThreadsPerBlock >= 128) {
        threadsPerBlock = 128;
    } else if (maxThreadsPerBlock >= 64) {
        threadsPerBlock = 64;
    } else if (maxThreadsPerBlock >= 32) {
        threadsPerBlock = 32;
    } else {
        threadsPerBlock = 16;
    }
    unsigned int numBlocks = (num_pixels + threadsPerBlock - 1) / threadsPerBlock;

    compute_sfrd_kernel<<< numBlocks, threadsPerBlock >>>(SFRD_conditional_table->x_min, SFRD_conditional_table->x_width, d_y_arr, d_dens_R_grid, zpp_growth_R_ct, d_sfrd_grid, d_fcoll_tmp, num_pixels);
    // CALL_CUDA(hipDeviceSynchronize());
    CALL_CUDA(hipGetLastError());
    LOG_INFO("SpinTemperatureBox compute_sfrd_kernel called.");

    // Copy results from device to host
    CALL_CUDA(hipMemcpy(sfrd_grid, d_sfrd_grid, sizeof(float) * num_pixels, hipMemcpyDeviceToHost));
    LOG_INFO("SFRD sum copied to host.");

    // Use thrust to sum temp fcoll array for ave_sfrd
    // Wrap device pointer in a thrust::device_ptr
    thrust::device_ptr<double> d_fcoll_tmp_ptr(d_fcoll_tmp);
    // Sum temp fcoll array
    double sum_sfrd = thrust::reduce(d_fcoll_tmp_ptr, d_fcoll_tmp_ptr + num_pixels, 0., thrust::plus<double>());
    CALL_CUDA(hipGetLastError());
    LOG_INFO("SFRD sum reduced to single value by thrust::reduce operation.");

    return sum_sfrd;
}

void free_sfrd_gpu_data_simple(
    float **d_y_arr, // copies of pointers to pointers
    float **d_dens_R_grid,
    float **d_sfrd_grid,
    double **d_fcoll_tmp
) {
    // Need to dereference the pointers to pointers (*)
    CALL_CUDA(hipFree(*d_y_arr));
    CALL_CUDA(hipFree(*d_dens_R_grid));
    CALL_CUDA(hipFree(*d_sfrd_grid));
    CALL_CUDA(hipFree(*d_fcoll_tmp));
    LOG_INFO("Device memory freed.");
}
