#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <math.h>
#include <omp.h>
#include <complex.h>
#include <fftw3.h>
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>
#include "cexcept.h"
#include "exceptions.h"
#include "logger.h"

// GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
// We use thrust for reduction
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/functional.h> // thrust::plus

#include "Constants.h"
#include "InputParameters.h"
#include "OutputStructs.h"
#include "cosmology.h"
#include "hmf.h"
#include "indexing.h"
#include "dft.h"
#include "recombinations.h"
#include "debugging.h"
#include "heating_helper_progs.h"
#include "photoncons.h"
#include "thermochem.h"
#include "interp_tables.h"
#include "filtering.h"
#include "bubble_helper_progs.h"
#include "InitialConditions.h"

#include "IonisationBox.h"
#include "cuda_utils.cuh"


__device__ inline double EvaluateRGTable1D_f_gpu(double x, double x_min, double x_width, float *y_arr) {

    int idx = (int)floor((x - x_min) / x_width);

    double table_val = x_min + x_width * (float)idx;
    double interp_point = (x - table_val) / x_width;

    return y_arr[idx] * (1 - interp_point) + y_arr[idx + 1] * (interp_point);
}

// template <unsigned int threadsPerBlock>
__global__ void compute_Fcoll(
    hipFloatComplex *deltax_filtered, // fg_struct
    hipFloatComplex *xe_filtered, // fg_struct
    float *y_arr, // Nion_conditional_table1D
    double x_min, // Nion_conditional_table1D
    double x_width, // Nion_conditional_table1D
    double fract_float_err, // FRACT_FLOAT_ERR
    bool use_ts_fluct, // flag_options_global->USE_TS_FLUCT
    unsigned long long hii_tot_num_pixels, // HII_TOT_NUM_PIXELS
    long long hii_d, // HII_D
    long long hii_d_para, // HII_D_PARA
    long long hii_mid_para, // HII_MID_PARA
    float *Fcoll // box
) {
    // Get index of grids
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Bound check
    if (idx >= hii_tot_num_pixels) {
        return;
    }

    // Get x, y, z from idx using HII_R_INDEX macro formula
    int z = idx % hii_d_para;
    unsigned long long remaining = idx / hii_d_para;
    int y = remaining % hii_d;
    int x = remaining / hii_d;

    // Get FFT index using HII_R_FFT_INDEX macro formula
    unsigned long long fft_idx = z + 2 * (hii_mid_para + 1) * (y + hii_d * x);

    // These clippings could be made in the calling function, using thrust, rather than here...

    // Clip the filtered grids to physical values
    // delta cannot be less than -1
    *((float *) deltax_filtered + fft_idx) = fmaxf(*((float *) deltax_filtered + fft_idx), -1. + fract_float_err);
    // <N_rec> cannot be less than zero
    // x_e has to be between zero and unity
    if (use_ts_fluct) {
        *((float *) xe_filtered + fft_idx) = fmaxf(*((float *) xe_filtered + fft_idx), 0.0);
        *((float *) xe_filtered + fft_idx) = fminf(*((float *) xe_filtered + fft_idx), 0.999);
    }

    // Compute collapse fraction
    Fcoll[idx] = exp(EvaluateRGTable1D_f_gpu(*((float *) deltax_filtered + fft_idx), x_min, x_width, y_arr));
}

void init_ionbox_gpu_data(
    fftwf_complex **d_deltax_filtered, // copies of pointers to pointers
    fftwf_complex **d_xe_filtered,
    float **d_y_arr,
    float **d_Fcoll,
    unsigned int nbins, // nbins for Nion_conditional_table1D->y
    unsigned long long hii_tot_num_pixels, // HII_TOT_NUM_PIXELS
    unsigned long long hii_kspace_num_pixels, // HII_KSPACE_NUM_PIXELS
    unsigned int *threadsPerBlock,
    unsigned int *numBlocks
) {
    CALL_CUDA(hipMalloc((void**)d_deltax_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels)); // already pointers to pointers (no & needed)
    CALL_CUDA(hipMemset(*d_deltax_filtered, 0, sizeof(fftwf_complex) * hii_kspace_num_pixels)); // dereference the pointer to a pointer (*)

    if (flag_options_global->USE_TS_FLUCT) {
        CALL_CUDA(hipMalloc((void**)d_xe_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels));
        CALL_CUDA(hipMemset(*d_xe_filtered, 0, sizeof(fftwf_complex) * hii_kspace_num_pixels));
    }

    CALL_CUDA(hipMalloc((void**)d_y_arr, sizeof(float) * nbins));
    CALL_CUDA(hipMemset(*d_y_arr, 0, sizeof(float) * nbins));

    CALL_CUDA(hipMalloc((void**)d_Fcoll, sizeof(float) * hii_tot_num_pixels));
    CALL_CUDA(hipMemset(*d_Fcoll, 0, sizeof(float) * hii_tot_num_pixels));

    LOG_INFO("Ionisation grids allocated on device.");
    LOG_INFO("Ionisation grids initialised on device.");

    // Get max threads/block for device
    int maxThreadsPerBlock;
    CALL_CUDA(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0));

    // Set threads/block based on device max
    if (maxThreadsPerBlock >= 512) {
        *threadsPerBlock = 512;
    } else if (maxThreadsPerBlock >= 256) {
        *threadsPerBlock = 256;
    } else if (maxThreadsPerBlock >= 128) {
        *threadsPerBlock = 128;
    } else if (maxThreadsPerBlock >= 64) {
        *threadsPerBlock = 64;
    } else if (maxThreadsPerBlock >= 32) {
        *threadsPerBlock = 32;
    } else {
        *threadsPerBlock = 16;
    }

    *numBlocks = (hii_tot_num_pixels + *threadsPerBlock - 1) / *threadsPerBlock;
}

void calculate_fcoll_grid_gpu(
    IonizedBox *box, // for box->Fcoll
    fftwf_complex *h_deltax_filtered, // members of fg_struct
    fftwf_complex *h_xe_filtered,
    double *f_coll_grid_mean, // member of rspec
    fftwf_complex *d_deltax_filtered, // device pointers
    fftwf_complex *d_xe_filtered,
    float *d_Fcoll,
    float *d_y_arr,
    unsigned long long hii_tot_num_pixels, // HII_TOT_NUM_PIXELS
    unsigned long long hii_kspace_num_pixels, // HII_KSPACE_NUM_PIXELS
    unsigned int *threadsPerBlock,
    unsigned int *numBlocks
) {
    RGTable1D_f* Nion_conditional_table1D = get_Nion_conditional_table1D();

    // Copy grids from host to device
    CALL_CUDA(hipMemcpy(d_deltax_filtered, h_deltax_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels, hipMemcpyHostToDevice));
    if (flag_options_global->USE_TS_FLUCT) {
        CALL_CUDA(hipMemcpy(d_xe_filtered, h_xe_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels, hipMemcpyHostToDevice));
    }
    CALL_CUDA(hipMemcpy(d_y_arr, Nion_conditional_table1D->y_arr, sizeof(float) * Nion_conditional_table1D->n_bin, hipMemcpyHostToDevice));
    LOG_INFO("Ionisation grids copied to device.");

    // TODO: Can I pass these straight to kernel? (or access in kernel w/ Tiger's method)
    double fract_float_err = FRACT_FLOAT_ERR;
    bool use_ts_fluct = flag_options_global->USE_TS_FLUCT;
    long long hii_d = HII_D;
    long long hii_d_para = HII_D_PARA;
    long long hii_mid_para = HII_MID_PARA;

    // Invoke kernel
    compute_Fcoll<<< *numBlocks, *threadsPerBlock >>>(
        reinterpret_cast<hipFloatComplex *>(d_deltax_filtered),
        reinterpret_cast<hipFloatComplex *>(d_xe_filtered),
        d_y_arr,
        Nion_conditional_table1D->x_min,
        Nion_conditional_table1D->x_width,
        fract_float_err,
        use_ts_fluct,
        hii_tot_num_pixels,
        hii_d,
        hii_d_para,
        hii_mid_para,
        d_Fcoll
    );
    CALL_CUDA(hipDeviceSynchronize());
    LOG_INFO("IonisationBox compute_Fcoll kernel called.");

    // Use thrust to reduce computed sums to one value.
    // Wrap device pointer in a thrust::device_ptr
    thrust::device_ptr<float> d_Fcoll_ptr(d_Fcoll);
    // Reduce final buffer sums to one value
    double f_coll_grid_total = thrust::reduce(d_Fcoll_ptr, d_Fcoll_ptr + hii_tot_num_pixels, 0., thrust::plus<float>());
    *f_coll_grid_mean = f_coll_grid_total / (double) hii_tot_num_pixels;
    LOG_INFO("Fcoll sum reduced to single value by thrust::reduce operation.");

    // Copy results from device to host
    CALL_CUDA(hipMemcpy(box->Fcoll, d_Fcoll, sizeof(float) * hii_tot_num_pixels, hipMemcpyDeviceToHost));
    CALL_CUDA(hipMemcpy(h_deltax_filtered, d_deltax_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels, hipMemcpyDeviceToHost));
    if (flag_options_global->USE_TS_FLUCT) {
        CALL_CUDA(hipMemcpy(h_xe_filtered, d_xe_filtered, sizeof(fftwf_complex) * hii_kspace_num_pixels, hipMemcpyDeviceToHost));
    }
    LOG_INFO("Grids copied to host.");
}

void free_ionbox_gpu_data(
    fftwf_complex **d_deltax_filtered, // copies of pointers to pointers
    fftwf_complex **d_xe_filtered,
    float **d_y_arr,
    float **d_Fcoll
) {
    CALL_CUDA(hipFree(*d_deltax_filtered)); // Need to dereference the pointers to pointers (*)
    if (flag_options_global->USE_TS_FLUCT) {
        CALL_CUDA(hipFree(*d_xe_filtered));
    }
    CALL_CUDA(hipFree(*d_y_arr));
    CALL_CUDA(hipFree(*d_Fcoll));
    LOG_INFO("Device memory freed.");
}
